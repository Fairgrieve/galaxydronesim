﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "PlanetarySystem.h"
#include "StarSystem.h"
#include <stdio.h>

// This function sets up the CUDA environment and allocates memory.
hipError_t initializeCuda();

// Main function
int main()
{
    hipError_t cudaStatus = initializeCuda();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "initializeCuda failed!");
        return 1;
    }

    // Create some planets
    Planet earth("Earth", 5.972e24, 1.496e8, 0.0, 0.0, 29.78);
    Planet mars("Mars", 0.64171e24, 2.279e8, 0.0, 0.0, 24.077);

    // Create some stars
    Star sun("Sun", 1.989e30, 0.0, 0.0, 0.0, 0.0, 5778, 1.0, 1.0, 4.603e9, 'G');
    Star sirius("Sirius", 2.063e30, 8.6e16, 0.0, 0.0, 0.0, 9940, 1.711, 25.4, 2.5e8, 'A');

    // Output the names of the planets and stars
    printf("Planet 1: %s\n", earth.getName().c_str());
    printf("Planet 2: %s\n", mars.getName().c_str());
    printf("Star 1: %s\n", sun.getName().c_str());
    printf("Star 2: %s\n", sirius.getName().c_str());

    // Test updating position for planets
    double timeStep = 1.0; // 1 second
    earth.updatePosition(timeStep);
    mars.updatePosition(timeStep);
    printf("After 1 second:\n");
    printf("Earth position: (%.2f, %.2f)\n", earth.getXPos(), earth.getYPos());
    printf("Mars position: (%.2f, %.2f)\n", mars.getXPos(), mars.getYPos());

    // Test updating position for stars
    sun.updatePosition(timeStep);
    sirius.updatePosition(timeStep);
    printf("After 1 second:\n");
    printf("Sun position: (%.2f, %.2f)\n", sun.getXPos(), sun.getYPos());
    printf("Sirius position: (%.2f, %.2f)\n", sirius.getXPos(), sirius.getYPos());

    // Test setting new position and velocity for planets
    earth.setPosition(1.5e8, 0.0);
    earth.setVelocity(0.0, 29.78);
    mars.setPosition(2.3e8, 0.0);
    mars.setVelocity(0.0, 24.077);
    printf("After setting new position and velocity for planets:\n");
    printf("Earth new position: (%.2f, %.2f)\n", earth.getXPos(), earth.getYPos());
    printf("Earth new velocity: (%.2f, %.2f)\n", earth.getXVel(), earth.getYVel());
    printf("Mars new position: (%.2f, %.2f)\n", mars.getXPos(), mars.getYPos());
    printf("Mars new velocity: (%.2f, %.2f)\n", mars.getXVel(), mars.getYVel());

    // Test setting new position and velocity for stars
    sun.setPosition(0.0, 0.0);
    sun.setVelocity(0.0, 0.0);
    sirius.setPosition(8.6e16, 0.0);
    sirius.setVelocity(0.0, 0.0);
    printf("After setting new position and velocity for stars:\n");
    printf("Sun new position: (%.2f, %.2f)\n", sun.getXPos(), sun.getYPos());
    printf("Sun new velocity: (%.2f, %.2f)\n", sun.getXVel(), sun.getYVel());
    printf("Sirius new position: (%.2f, %.2f)\n", sirius.getXPos(), sirius.getYPos());
    printf("Sirius new velocity: (%.2f, %.2f)\n", sirius.getXVel(), sirius.getYVel());

    // Reset the device and free memory before exiting
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for initializing CUDA and allocating memory.
hipError_t initializeCuda()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    // Allocate GPU buffers (this is just a placeholder, you'll need actual buffers for your project)
    // Example:
    // int *dev_buffer;
    // cudaStatus = hipMalloc((void**)&dev_buffer, bufferSize * sizeof(int));
    // if (cudaStatus != hipSuccess) {
    //     fprintf(stderr, "hipMalloc failed!");
    //     return cudaStatus;
    // }

    return cudaStatus;
}

